#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <iostream>

#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/Parallel.h>

// #include "utils.cuh"

#include <Eigen/Sparse>
#include <Eigen/SparseCore>
#include <Eigen/SparseCholesky>

typedef Eigen::SparseMatrix<double> SpMat;
typedef Eigen::Triplet<double> T;
typedef std::vector<std::vector<long>> graph_t;
typedef std::vector<torch::Tensor> tensor_list_t;



#define MIN_DEPTH 0.25

#define THREADS 256
#define NUM_BLOCKS(batch_size) ((batch_size + THREADS - 1) / THREADS)


#define GPU_1D_KERNEL_LOOP(k, n) \
  for (size_t k = threadIdx.x; k<n; k += blockDim.x)


__device__ void warpReduce(volatile float *sdata, unsigned int tid) {
  sdata[tid] += sdata[tid + 32];
  sdata[tid] += sdata[tid + 16];
  sdata[tid] += sdata[tid +  8];
  sdata[tid] += sdata[tid +  4];
  sdata[tid] += sdata[tid +  2];
  sdata[tid] += sdata[tid +  1];
}

__device__ void blockReduce(volatile float *sdata) {
  unsigned int tid = threadIdx.x;
  __syncthreads();

  // if (threadIdx.x < 256) {sdata[tid] += sdata[tid + 256]; } __syncthreads();
  if (threadIdx.x < 128) {sdata[tid] += sdata[tid + 128]; } __syncthreads();
  if (threadIdx.x <  64) {sdata[tid] += sdata[tid +  64]; } __syncthreads();

  if (tid < 32) warpReduce(sdata, tid);
  __syncthreads();
}


//    EIGEN_DEVICE_FUNC Point operator*(Point const& p) const {
//      const Quaternion& q = unit_quaternion;
//      Point uv = q.vec().cross(p);
//      uv += uv;
//      return p + q.w()*uv + q.vec().cross(uv);
//    }
// Mind that sometimes I pass Y == X, so don't use X[i] after editing Y[i]
// since they might be at the same memory location...
__device__ void
actSO3(const float *q, const float *X, float *Y) {
  float uv[3];
  uv[0] = 2.0 * (q[1]*X[2] - q[2]*X[1]);
  uv[1] = 2.0 * (q[2]*X[0] - q[0]*X[2]);
  uv[2] = 2.0 * (q[0]*X[1] - q[1]*X[0]);

  Y[0] = X[0] + q[3]*uv[0] + (q[1]*uv[2] - q[2]*uv[1]);
  Y[1] = X[1] + q[3]*uv[1] + (q[2]*uv[0] - q[0]*uv[2]);
  Y[2] = X[2] + q[3]*uv[2] + (q[0]*uv[1] - q[1]*uv[0]);
}

__device__  void
actSE3(const float *t, const float *q, const float *X, float *Y) {
  actSO3(q, X, Y);
  Y[3] = X[3];
  Y[0] += X[3] * t[0];
  Y[1] += X[3] * t[1];
  Y[2] += X[3] * t[2];
}

// Sometimes we send X == Y
__device__ void
adjSE3(const float *t, const float *q, const float *X, float *Y) {
  float qinv[4] = {-q[0], -q[1], -q[2], q[3]};
  actSO3(qinv, &X[0], &Y[0]); // Calculates [X0, X1, X2] * R_ij = (R_ij^t *[X0,X1,X2]^t)^t (upper left of Ji * Adj_ij)
  actSO3(qinv, &X[3], &Y[3]); // Calculates [X3, X4, X5] * R_ij (lower right of Ji * Adj_ij)

  // Calculates [X0, X1, X3] * [t]_x (aka cross product of X0..3 and t)
  float u[3], v[3]; 
  u[0] = t[2]*X[1] - t[1]*X[2];
  u[1] = t[0]*X[2] - t[2]*X[0];
  u[2] = t[1]*X[0] - t[0]*X[1];

  // Last rotation multiplication, and obtain upper right of Ji * Adj_ij
  actSO3(qinv, u, v);
  Y[3] += v[0];
  Y[4] += v[1];
  Y[5] += v[2];
}

// Computes: Gi, Gj -> Gij = Gj * Gi^{-1}
__device__ void 
relSE3(const float *ti, const float *qi, const float *tj, const float *qj, float *tij, float *qij) {
  // Hamilton product of two quaternions, qj is inverted. TODO: check this math...
  qij[0] = -qj[3] * qi[0] + qj[0] * qi[3] - qj[1] * qi[2] + qj[2] * qi[1],
  qij[1] = -qj[3] * qi[1] + qj[1] * qi[3] - qj[2] * qi[0] + qj[0] * qi[2],
  qij[2] = -qj[3] * qi[2] + qj[2] * qi[3] - qj[0] * qi[1] + qj[1] * qi[0], 
  qij[3] =  qj[3] * qi[3] + qj[0] * qi[0] + qj[1] * qi[1] + qj[2] * qi[2], 

  actSO3(qij, ti, tij);
  tij[0] = tj[0] - tij[0];
  tij[1] = tj[1] - tij[1];
  tij[2] = tj[2] - tij[2];
}

  
__device__ void
expSO3(const float *phi, float* q) {
  // SO3 exponential map
  float theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
  float theta_p4 = theta_sq * theta_sq;

  float theta = sqrtf(theta_sq);
  float imag, real;

  if (theta_sq < 1e-8) {
    imag = 0.5 - (1.0/48.0)*theta_sq + (1.0/3840.0)*theta_p4;
    real = 1.0 - (1.0/ 8.0)*theta_sq + (1.0/ 384.0)*theta_p4;
  } else {
    imag = sinf(0.5 * theta) / theta;
    real = cosf(0.5 * theta);
  }

  q[0] = imag * phi[0];
  q[1] = imag * phi[1];
  q[2] = imag * phi[2];
  q[3] = real;

}

__device__ void
crossInplace(const float* a, float *b) {
  float x[3] = {
    a[1]*b[2] - a[2]*b[1],
    a[2]*b[0] - a[0]*b[2],
    a[0]*b[1] - a[1]*b[0], 
  };

  b[0] = x[0];
  b[1] = x[1];
  b[2] = x[2];
}

__device__ void
expSE3(const float *xi, float* t, float* q) {
  // SE3 exponential map

  expSO3(xi + 3, q);
  float tau[3] = {xi[0], xi[1], xi[2]};
  float phi[3] = {xi[3], xi[4], xi[5]};

  float theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
  float theta = sqrtf(theta_sq);

  t[0] = tau[0]; 
  t[1] = tau[1]; 
  t[2] = tau[2];

  if (theta > 1e-4) {
    float a = (1 - cosf(theta)) / theta_sq;
    crossInplace(phi, tau);
    t[0] += a * tau[0];
    t[1] += a * tau[1];
    t[2] += a * tau[2];

    float b = (theta - sinf(theta)) / (theta * theta_sq);
    crossInplace(phi, tau);
    t[0] += b * tau[0];
    t[1] += b * tau[1];
    t[2] += b * tau[2];
  }
}


// Doesn't have concept of kf0, kf1! Flows are hence computed outside optimization window as well.
__global__ void projective_transform_kernel(
    const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> target,
    const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> weight,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> body_poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> extrinsics, // cam_T_imu
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> Hs,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> vs,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> Eiz,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> Ejz,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> Cii,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> bz)
{
  const int M = blockIdx.x; // Number of flow measurements.
  const int thread_id = threadIdx.x;

  const int ht = disps.size(1);
  const int wd = disps.size(2);

  int ix = static_cast<int>(ii[M]);
  int jx = static_cast<int>(jj[M]);

  __shared__ float fx;
  __shared__ float fy;
  __shared__ float cx;
  __shared__ float cy;

  __shared__ float ti[3], tj[3], tij[3];
  __shared__ float qi[4], qj[4], qij[4];

  __shared__ float cam_T_body_tij[3];
  __shared__ float cam_T_body_qij[4];

  // load intrinsics from global memory
  if (thread_id == 0) {
    fx = intrinsics[0];
    fy = intrinsics[1];
    cx = intrinsics[2];
    cy = intrinsics[3];

    // Hard-code extrinsics for now! cam_T_imu
    cam_T_body_tij[0] = extrinsics[0]; // 0.0652229;
    cam_T_body_tij[1] = extrinsics[1]; // -0.0207064;
    cam_T_body_tij[2] = extrinsics[2]; // -0.0080546;

    cam_T_body_qij[0] = extrinsics[3]; // 0.00770718; //x
    cam_T_body_qij[1] = extrinsics[4]; // -0.01049932;//y
    cam_T_body_qij[2] = extrinsics[5]; // -0.7017528; //z
    cam_T_body_qij[3] = extrinsics[6]; // 0.71230146; //w
  }

  __syncthreads();

  if (ix == jx) {
    // stereo frames
    if (thread_id == 0) {
      tij[0] =  -0.1; // Baseline...
      tij[1] =     0;
      tij[2] =     0;
      qij[0] =     0;
      qij[1] =     0;
      qij[2] =     0;
      qij[3] =     1;
    }
  } else {
    // mono frames

    // load poses from global memory
    if (thread_id < 3) {
      ti[thread_id] = poses[ix][thread_id];
      tj[thread_id] = poses[jx][thread_id];
    }

    if (thread_id < 4) {
      qi[thread_id] = poses[ix][thread_id+3];
      qj[thread_id] = poses[jx][thread_id+3];
    }

    __syncthreads();

    if (thread_id == 0) {
      relSE3(ti, qi, tj, qj, tij, qij);
    }
  }

  __syncthreads();

  //points 
  float Xi[4];
  float Xj[4];

  // jacobians
  float Jx[12];
  float Jz;

  float* Ji = &Jx[0];
  float* Jj = &Jx[6];

  // hessians
  // 12*(12+1)/2 = 78
  // These are the upper triangular elements of the hessian matrix contributed by the point to the camera blocks.
  // H = [Hii Hij; Hji Hjj], where Hii and Hjj are symmetric. The actual hessian is the summation of all these blocks from the points.
  float hij[12*(12+1)/2]; // Upper triangular part of the Hessian only.

  float vi[6], vj[6];

  // Init hij's values to zero...
  int l;
  for (l=0; l<12*(12+1)/2; l++) {
    hij[l] = 0;
  }

  // Init vi, vj to zero.
  for (int n=0; n<6; n++) {
    vi[n] = 0;
    vj[n] = 0;
  }

  __syncthreads();

  // AKA: for (size_t k = threadIdx.x; k<n; k += blockDim.x)
  GPU_1D_KERNEL_LOOP(k, ht*wd) { // AKA: loops over all pixels in the image by dividing chunks of pixels to threads.

    const int i = k / wd;
    const int j = k % wd;

    // We are at pixel (u,v) in the image of keyframe ix, and flow goes from ix to jx
    const float u = static_cast<float>(j);
    const float v = static_cast<float>(i);
    
    // homogenous coordinates
    Xi[0] = (u - cx) / fx;
    Xi[1] = (v - cy) / fy;
    Xi[2] = 1;
    Xi[3] = disps[ix][i][j]; // ix is the keyframe index.

    // transform homogenous point from keyframe ix to keyframe jx, for pixel (u,v)
    actSE3(tij, qij, Xi, Xj);

    const float x = Xj[0];
    const float y = Xj[1];
    const float h = Xj[3];

    const float d = (Xj[2] < MIN_DEPTH) ? 0.0 : 1.0 / Xj[2];
    const float d2 = d * d;

    // Get the weight of the pixel (u,v)==(j,i) for the flow measurement number `block_id'
    // TODO: why the 0.001 !? Shouldn't it be Xj[3]?
    float weight_u = (Xj[2] < MIN_DEPTH) ? 0.0 : .001 * weight[M][0][i][j];
    float weight_v = (Xj[2] < MIN_DEPTH) ? 0.0 : .001 * weight[M][1][i][j];

    // Calculate the difference between the expected and current flow
    const float residual_u = target[M][0][i][j] - (fx * d * x + cx);
    const float residual_v = target[M][1][i][j] - (fy * d * y + cy);

    // Populate hessian and information vectors with jacobians
    // [H  Eii/Eij]
    // [   Cii]
    ////////////// x - coordinate 

    // x - coordinate of D(p')/D(psi_j)

    // Jacobian of reprojection function of u-coord of pixel wrt depth Z = 1/d
    // 1/d = Z; d2 = 1/Z^2; x = X; tij[0] = tx; tij[2] = tz;
    // Jz = fx * 1/Z * tx - fx * X/Z^2 * tz
    Jz = fx * (tij[0] * d - tij[2] * (x * d2)); // this is a scalar...

    // The C block is stored as a MxHW matrix, where M is the number of measurements, and HW is the number of pixels.
    Cii[M][k] = weight_u * Jz * Jz;
    bz[M][k] = weight_u * residual_u * Jz;

    if (ix == jx) weight_u = 0; // for stereo

    Jj[0] = fx * (h*d);
    Jj[1] = fx * 0.0;
    Jj[2] = fx * (-x*h*d2);
    Jj[3] = fx * (-x*y*d2);
    Jj[4] = fx * (1.0 + x*x*d2);
    Jj[5] = fx * (-y*d);

    adjSE3(tij, qij, Jj, Ji);
    for (int n=0; n<6; n++) Ji[n] *= -1.0;

    // if cam_T_body:
    adjSE3(cam_T_body_tij, cam_T_body_qij, Jj, Jj);
    adjSE3(cam_T_body_tij, cam_T_body_qij, Ji, Ji);

    // To get right jacobians wrt world_T_body
    for (int n=0; n<6; n++) Jj[n] *= -1.0;
    for (int n=0; n<6; n++) Ji[n] *= -1.0;

    // To get [wx wy wz tx ty tz] (gtsam) instead of [tx ty tz wx wy wz] (droid)
    float Jj_copy[6];
    for (int n=0; n<6; n++) Jj_copy[n] = Jj[n];
    Jj[0] = Jj_copy[3];
    Jj[1] = Jj_copy[4];
    Jj[2] = Jj_copy[5];
    Jj[3] = Jj_copy[0];
    Jj[4] = Jj_copy[1];
    Jj[5] = Jj_copy[2];
    float Ji_copy[6];
    for (int n=0; n<6; n++) Ji_copy[n] = Ji[n];
    Ji[0] = Ji_copy[3];
    Ji[1] = Ji_copy[4];
    Ji[2] = Ji_copy[5];
    Ji[3] = Ji_copy[0];
    Ji[4] = Ji_copy[1];
    Ji[5] = Ji_copy[2];

    // Compute contributions to the Hessian for the pose to pose block
    l=0;
    for (int n=0; n<12; n++) {
      for (int m=0; m<=n; m++) { // we only compute the upper diagonal, so stop at m at n...
        hij[l] += weight_u * Jx[n] * Jx[m];
        l++;
      }
    }

    for (int n=0; n<6; n++) {
      // weight_u = 0 for stereo
      vi[n] += weight_u * residual_u * Ji[n];
      vj[n] += weight_u * residual_u * Jj[n];

      Eiz[M][n][k] = weight_u * Jz * Ji[n]; // Jz is a scalar
      Ejz[M][n][k] = weight_u * Jz * Jj[n];
    }

    ////////////// y - coordinate of D(p')/D(psi_j)

    // TODO: Parallelize the calculations above and below this line...

    // y - coordinate of D(p')/D(psi_j)
    Jz = fy * (tij[1] * d - tij[2] * (y * d2));
    Cii[M][k] += weight_v * Jz * Jz;
    bz[M][k] += weight_v * residual_v * Jz;

    if (ix == jx) weight_v = 0; // For stereo

    Jj[0] = fy * 0;
    Jj[1] = fy * (h*d);
    Jj[2] = fy * (-y*h*d2);
    Jj[3] = fy * (-1 - y*y*d2);
    Jj[4] = fy * (x*y*d2);
    Jj[5] = fy * (x*d);

    adjSE3(tij, qij, Jj, Ji);
    for (int n=0; n<6; n++) Ji[n] *= -1.0;

    // if cam_T_body:
    adjSE3(cam_T_body_tij, cam_T_body_qij, Jj, Jj);
    adjSE3(cam_T_body_tij, cam_T_body_qij, Ji, Ji);

    // To get right jacobians wrt world_T_body
    for (int n=0; n<6; n++) Jj[n] *= -1.0;
    for (int n=0; n<6; n++) Ji[n] *= -1.0;

    // To get [wx wy wz tx ty tz] (gtsam) instead of [tx ty tz wx wy wz] (droid)
    for (int n=0; n<6; n++) Jj_copy[n] = Jj[n];
    Jj[0] = Jj_copy[3];
    Jj[1] = Jj_copy[4];
    Jj[2] = Jj_copy[5];
    Jj[3] = Jj_copy[0];
    Jj[4] = Jj_copy[1];
    Jj[5] = Jj_copy[2];
    for (int n=0; n<6; n++) Ji_copy[n] = Ji[n];
    Ji[0] = Ji_copy[3];
    Ji[1] = Ji_copy[4];
    Ji[2] = Ji_copy[5];
    Ji[3] = Ji_copy[0];
    Ji[4] = Ji_copy[1];
    Ji[5] = Ji_copy[2];

    // Compute contributions to the Hessian
    l=0;
    for (int n=0; n<12; n++) {
      for (int m=0; m<=n; m++) {
        hij[l] += weight_v * Jx[n] * Jx[m];
        l++;
      }
    }

    for (int n=0; n<6; n++) {
      // weight_v = 0 for stereo
      vi[n] += weight_v * residual_v * Ji[n];
      vj[n] += weight_v * residual_v * Jj[n];

      // Cross-contributions to the Hessian for the pose and depth blocks
      // H = [B, E; Et, C], these are the E contributions for pixel (u,v) of flow from i to j.
      // The size of the matrices are M*D*HW, where M are the number of measurements, D the dimensionality of the pose,
      Eiz[M][n][k] += weight_v * Jz * Ji[n];
      Ejz[M][n][k] += weight_v * Jz * Jj[n];
    }

    // Done with per-pixel update?
  }

  __syncthreads();

  __shared__ float sdata[THREADS];
  for (int n=0; n<6; n++) {
    sdata[threadIdx.x] = vi[n];
    blockReduce(sdata);
    if (threadIdx.x == 0) {
      vs[0][M][n] = sdata[0];
    }

    __syncthreads();

    sdata[threadIdx.x] = vj[n];
    blockReduce(sdata);
    if (threadIdx.x == 0) {
      vs[1][M][n] = sdata[0];
    }

  }

  l=0;
  for (int n=0; n<12; n++) {
    for (int m=0; m<=n; m++) {
      sdata[threadIdx.x] = hij[l];
      blockReduce(sdata);

      if (threadIdx.x == 0) {
        if (n<6 && m<6) {
          Hs[0][M][n][m] = sdata[0];
          Hs[0][M][m][n] = sdata[0];
        }
        else if (n >=6 && m<6) {
          Hs[1][M][m][n-6] = sdata[0];
          Hs[2][M][n-6][m] = sdata[0];
        }
        else {
          Hs[3][M][n-6][m-6] = sdata[0];
          Hs[3][M][m-6][n-6] = sdata[0];
        }
      }

      l++;
    }
  }
}


__global__ void projmap_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> coords,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> valid)
{

  const int block_id = blockIdx.x;
  const int thread_id = threadIdx.x;

  const int ht = disps.size(1);
  const int wd = disps.size(2);

  __shared__ int ix;
  __shared__ int jx;

  __shared__ float fx;
  __shared__ float fy;
  __shared__ float cx;
  __shared__ float cy;

  __shared__ float ti[3], tj[3], tij[3];
  __shared__ float qi[4], qj[4], qij[4];

  // load intrinsics from global memory
  if (thread_id == 0) {
    ix = static_cast<int>(ii[block_id]);
    jx = static_cast<int>(jj[block_id]);
    fx = intrinsics[0];
    fy = intrinsics[1];
    cx = intrinsics[2];
    cy = intrinsics[3];
  }

  __syncthreads();

  // load poses from global memory
  if (thread_id < 3) {
    ti[thread_id] = poses[ix][thread_id];
    tj[thread_id] = poses[jx][thread_id];
  }

  if (thread_id < 4) {
    qi[thread_id] = poses[ix][thread_id+3];
    qj[thread_id] = poses[jx][thread_id+3];
  }

  __syncthreads();

  if (thread_id == 0) {
    relSE3(ti, qi, tj, qj, tij, qij);
  }

  //points 
  float Xi[4];
  float Xj[4];

  __syncthreads();

  GPU_1D_KERNEL_LOOP(k, ht*wd) {
    const int i = k / wd;
    const int j = k % wd;

    const float u = static_cast<float>(j);
    const float v = static_cast<float>(i);
    
    // homogenous coordinates
    Xi[0] = (u - cx) / fx;
    Xi[1] = (v - cy) / fy;
    Xi[2] = 1;
    Xi[3] = disps[ix][i][j];

    // transform homogenous point
    actSE3(tij, qij, Xi, Xj);

    coords[block_id][i][j][0] = u;
    coords[block_id][i][j][1] = v;

    if (Xj[2] > 0.01) {
      coords[block_id][i][j][0] = fx * (Xj[0] / Xj[2]) + cx;
      coords[block_id][i][j][1] = fy * (Xj[1] / Xj[2]) + cy;
    }

    valid[block_id][i][j][0] = (Xj[2] > MIN_DEPTH) ? 1.0 : 0.0;

  }
}

__global__ void frame_distance_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
    torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> dist,
    const float beta) {

  const int block_id = blockIdx.x;
  const int thread_id = threadIdx.x;

  const int ht = disps.size(1);
  const int wd = disps.size(2);

  __shared__ int ix;
  __shared__ int jx;

  __shared__ float fx;
  __shared__ float fy;
  __shared__ float cx;
  __shared__ float cy;

  __shared__ float ti[3], tj[3], tij[3];
  __shared__ float qi[4], qj[4], qij[4];

  // load intrinsics from global memory
  if (thread_id == 0) {
    ix = static_cast<int>(ii[block_id]);
    jx = static_cast<int>(jj[block_id]);
    fx = intrinsics[0];
    fy = intrinsics[1];
    cx = intrinsics[2];
    cy = intrinsics[3];
  }

  __syncthreads();


  //points 
  float Xi[4];
  float Xj[4];

  __shared__ float accum[THREADS]; accum[thread_id] = 0;
  __shared__ float valid[THREADS]; valid[thread_id] = 0;
  __shared__ float total[THREADS]; total[thread_id] = 0;

  __syncthreads();

  for (int n=0; n<1; n++) {

    if (thread_id < 3) {
      ti[thread_id] = poses[ix][thread_id];
      tj[thread_id] = poses[jx][thread_id];
    }

    if (thread_id < 4) {
      qi[thread_id] = poses[ix][thread_id+3];
      qj[thread_id] = poses[jx][thread_id+3];
    }

    __syncthreads();


    relSE3(ti, qi, tj, qj, tij, qij);

    float d, du, dv;

    GPU_1D_KERNEL_LOOP(k, ht*wd) {
      const int i = k / wd;
      const int j = k % wd;

      const float u = static_cast<float>(j);
      const float v = static_cast<float>(i);


      // if (disps[ix][i][j] < 0.01) {
      //   continue;
      // }
      
      // homogenous coordinates
      Xi[0] = (u - cx) / fx;
      Xi[1] = (v - cy) / fy;
      Xi[2] = 1;
      Xi[3] = disps[ix][i][j];

      // transform homogenous point
      actSE3(tij, qij, Xi, Xj);

      du = fx * (Xj[0] / Xj[2]) + cx - u;
      dv = fy * (Xj[1] / Xj[2]) + cy - v;
      d = sqrtf(du*du + dv*dv);

      total[threadIdx.x] += beta;
      
      if (Xj[2] > MIN_DEPTH) {
        accum[threadIdx.x] += beta * d;
        valid[threadIdx.x] += beta;
      }

      Xi[0] = (u - cx) / fx;
      Xi[1] = (v - cy) / fy;
      Xi[2] = 1;
      Xi[3] = disps[ix][i][j];

      Xj[0] = Xi[0] + Xi[3] * tij[0];
      Xj[1] = Xi[1] + Xi[3] * tij[1];
      Xj[2] = Xi[2] + Xi[3] * tij[2];

      du = fx * (Xj[0] / Xj[2]) + cx - u;
      dv = fy * (Xj[1] / Xj[2]) + cy - v;
      d = sqrtf(du*du + dv*dv);

      total[threadIdx.x] += (1 - beta);
      
      if (Xj[2] > MIN_DEPTH) {
        accum[threadIdx.x] += (1 - beta) * d;
        valid[threadIdx.x] += (1 - beta);
      }
    }

    if (threadIdx.x == 0) {
      int tmp = ix;
      ix = jx;
      jx = tmp;
    }

    __syncthreads();

  }
  __syncthreads(); blockReduce(accum);
  __syncthreads(); blockReduce(total);
  __syncthreads(); blockReduce(valid);

  __syncthreads();

  if (thread_id == 0) {
    dist[block_id] = (valid[0] / (total[0] + 1e-8) < 0.75) ? 1000.0 : accum[0] / valid[0];
  }
}



__global__ void depth_filter_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> inds,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> thresh,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> counter)
{

  const int block_id = blockIdx.x;
  const int neigh_id = blockIdx.y;
  const int index = blockIdx.z * blockDim.x + threadIdx.x;

  // if (threadIdx.x == 0) {
  //   printf("%d %d %d %d\n", blockIdx.x, blockIdx.y, blockDim.x, threadIdx.x);
  // }

  const int num = disps.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  __shared__ int ix;
  __shared__ int jx;

  __shared__ float fx;
  __shared__ float fy;
  __shared__ float cx;
  __shared__ float cy;

  __shared__ float ti[3], tj[3], tij[3];
  __shared__ float qi[4], qj[4], qij[4];

  if (threadIdx.x == 0) {
    ix = static_cast<int>(inds[block_id]);
    jx = (neigh_id < 3) ? ix - neigh_id - 1 : ix + neigh_id;
    fx = intrinsics[0];
    fy = intrinsics[1];
    cx = intrinsics[2];
    cy = intrinsics[3];
  }

  __syncthreads();

  if (jx < 0 || jx >= num) {
    return;
  }

  const float t = thresh[block_id];

  // load poses from global memory
  if (threadIdx.x < 3) {
    ti[threadIdx.x] = poses[ix][threadIdx.x];
    tj[threadIdx.x] = poses[jx][threadIdx.x];
  }

  if (threadIdx.x < 4) {
    qi[threadIdx.x] = poses[ix][threadIdx.x+3];
    qj[threadIdx.x] = poses[jx][threadIdx.x+3];
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    relSE3(ti, qi, tj, qj, tij, qij);
  }

  //points 
  float Xi[4];
  float Xj[4];

  __syncthreads();

  if (index < ht*wd) {
    const int i = index / wd;
    const int j = index % wd;

    const float ui = static_cast<float>(j);
    const float vi = static_cast<float>(i);
    const float di = disps[ix][i][j];
    
    // homogenous coordinates
    Xi[0] = (ui - cx) / fx;
    Xi[1] = (vi - cy) / fy;
    Xi[2] = 1;
    Xi[3] = di;

    // transform homogenous point
    actSE3(tij, qij, Xi, Xj);

    const float uj = fx * (Xj[0] / Xj[2]) + cx;
    const float vj = fy * (Xj[1] / Xj[2]) + cy;
    const float dj = Xj[3] / Xj[2];

    const int u0 = static_cast<int>(floor(uj));
    const int v0 = static_cast<int>(floor(vj));

    if (u0 >= 0 && v0 >= 0 && u0 < wd-1 && v0 < ht-1) {
      const float wx = ceil(uj) - uj;
      const float wy = ceil(vj) - vj;

      const float d00 = disps[jx][v0+0][u0+0];
      const float d01 = disps[jx][v0+0][u0+1];
      const float d10 = disps[jx][v0+1][u0+0];
      const float d11 = disps[jx][v0+1][u0+1];

      // err, dj_hat are not used, but it seems the idea was to compare the re-projected depth from i to j,
      // with the bilinear interpolated depth from pose j... And if they are closer than the thresh, than add +1
      // for the depth at pixel (i,j) and camera [block_id]
      const float dj_hat = wy*wx*d00 + wy*(1-wx)*d01 + (1-wy)*wx*d10 + (1-wy)*(1-wx)*d11;

      const float err = abs(1.0/dj - 1.0/dj_hat);

      // But then here it seems it decided to just count the number of neighbours close to the reprojected depth.
      if       (abs(1.0/dj - 1.0/d00) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
      else if  (abs(1.0/dj - 1.0/d01) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
      else if  (abs(1.0/dj - 1.0/d10) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
      else if  (abs(1.0/dj - 1.0/d11) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
    }
  }
}



__global__ void iproj_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> points)

{

  const int block_id = blockIdx.x;
  const int index = blockIdx.y * blockDim.x + threadIdx.x;


  const int num = disps.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  __shared__ float fx;
  __shared__ float fy;
  __shared__ float cx;
  __shared__ float cy;

  __shared__ float t[3];
  __shared__ float q[4];

  if (threadIdx.x == 0) {
    fx = intrinsics[0];
    fy = intrinsics[1];
    cx = intrinsics[2];
    cy = intrinsics[3];
  }

  __syncthreads();


  // load poses from global memory
  if (threadIdx.x < 3) {
    t[threadIdx.x] = poses[block_id][threadIdx.x];
  }

  if (threadIdx.x < 4) {
    q[threadIdx.x] = poses[block_id][threadIdx.x+3];
  }

  __syncthreads();

  //points 
  float Xi[4];
  float Xj[4];

  if (index < ht*wd) {
    const int i = index / wd;
    const int j = index % wd;

    const float ui = static_cast<float>(j);
    const float vi = static_cast<float>(i);
    const float di = disps[block_id][i][j];
    
    // homogenous coordinates
    Xi[0] = (ui - cx) / fx;
    Xi[1] = (vi - cy) / fy;
    Xi[2] = 1;
    Xi[3] = di;

    // transform homogenous point
    actSE3(t, q, Xi, Xj);

    points[block_id][i][j][0] = Xj[0] / Xj[3];
    points[block_id][i][j][1] = Xj[1] / Xj[3];
    points[block_id][i][j][2] = Xj[2] / Xj[3];

  }
}



__global__ void accum_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> inps,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ptrs,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> idxs,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> outs)
{
  
  const int block_id = blockIdx.x;
  const int D = inps.size(2);

  const int start = ptrs[block_id];
  const int end = ptrs[block_id+1];

  for (int k=threadIdx.x; k<D; k+=blockDim.x) {
    float x = 0;
    for (int i=start; i<end; i++) {
      x += inps[idxs[i]][k];
    }
    outs[block_id][k] = x;
  }  
}


__device__ void
retrSE3(const float *xi, const float* t, const float* q, float* t1, float* q1) {
  // retraction on SE3 manifold

  float dt[3] = {0, 0, 0};
  float dq[4] = {0, 0, 0, 1};
  
  expSE3(xi, dt, dq);

  q1[0] = dq[3] * q[0] + dq[0] * q[3] + dq[1] * q[2] - dq[2] * q[1];
  q1[1] = dq[3] * q[1] + dq[1] * q[3] + dq[2] * q[0] - dq[0] * q[2];
  q1[2] = dq[3] * q[2] + dq[2] * q[3] + dq[0] * q[1] - dq[1] * q[0];
  q1[3] = dq[3] * q[3] - dq[0] * q[0] - dq[1] * q[1] - dq[2] * q[2];

  actSO3(dq, t, t1);
  t1[0] += dt[0];
  t1[1] += dt[1];
  t1[2] += dt[2];
}


__global__ void pose_retr_kernel(
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> dx,
    const int kf0, const int kf1) 
{

  for (int k=kf0+threadIdx.x; k<kf1; k+=blockDim.x) {
    float xi[6], q[4], q1[4], t[3], t1[3];

    t[0] = poses[k][0];
    t[1] = poses[k][1];
    t[2] = poses[k][2];

    q[0] = poses[k][3];
    q[1] = poses[k][4];
    q[2] = poses[k][5];
    q[3] = poses[k][6];
    
    for (int n=0; n<6; n++) {
      xi[n] = dx[k-kf0][n];
    }

    retrSE3(xi, t, q, t1, q1);

    poses[k][0] = t1[0];
    poses[k][1] = t1[1];
    poses[k][2] = t1[2];

    poses[k][3] = q1[0];
    poses[k][4] = q1[1];
    poses[k][5] = q1[2];
    poses[k][6] = q1[3];
  }
}

__global__ void disp_retr_kernel(
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> dz,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> inds) 
{
  const int i = inds[blockIdx.x];
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  for (int k=threadIdx.x; k<ht*wd; k+=blockDim.x) {
    float d = disps[i][k/wd][k%wd] + dz[blockIdx.x][k];
    disps[i][k/wd][k%wd] = d;
  }
}

torch::Tensor accum_cuda(torch::Tensor data, torch::Tensor ix, torch::Tensor jx) {
  torch::Tensor ix_cpu = ix.to(torch::kCPU);
  torch::Tensor jx_cpu = jx.to(torch::kCPU);
  torch::Tensor inds = torch::argsort(ix_cpu);

  long* ix_data = ix_cpu.data_ptr<long>();
  long* jx_data = jx_cpu.data_ptr<long>();
  long* kx_data = inds.data_ptr<long>();

  int count = jx.size(0);
  std::vector<int> cols;

  torch::Tensor ptrs_cpu = torch::zeros({count+1}, 
    torch::TensorOptions().dtype(torch::kInt64));
  
  long* ptrs_data = ptrs_cpu.data_ptr<long>();
  ptrs_data[0] = 0;

  int i = 0;
  for (int j=0; j<count; j++) {
    while (i < ix.size(0) && ix_data[kx_data[i]] <= jx_data[j]) {
      if (ix_data[kx_data[i]] == jx_data[j])
        cols.push_back(kx_data[i]);
      i++;
    }
    ptrs_data[j+1] = cols.size();
  }

  torch::Tensor idxs_cpu = torch::zeros({long(cols.size())}, 
    torch::TensorOptions().dtype(torch::kInt64));

  long* idxs_data = idxs_cpu.data_ptr<long>();

  for (int i=0; i<cols.size(); i++) {
    idxs_data[i] = cols[i];
  }

  torch::Tensor ptrs = ptrs_cpu.to(torch::kCUDA);
  torch::Tensor idxs = idxs_cpu.to(torch::kCUDA);

  torch::Tensor out = torch::zeros({jx.size(0), data.size(1)},
    torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));

  accum_kernel<<<count, THREADS>>>(
    data.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    ptrs.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    idxs.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    out.packed_accessor32<float,2,torch::RestrictPtrTraits>());

  return out;
}


__global__ void EEt6x6_kernel(
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> E,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> Q,
    const torch::PackedTensorAccessor32<long,2,torch::RestrictPtrTraits> idx,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> S)
{

  // indicices
  const int ix = idx[blockIdx.x][0];
  const int jx = idx[blockIdx.x][1];
  const int kx = idx[blockIdx.x][2];

  const int D = E.size(2);

  float dS[6][6];
  float ei[6];
  float ej[6];

  for (int i=0; i<6; i++) {
    for (int j=0; j<6; j++) {
      dS[i][j] = 0;
    }
  }

  for (int k=threadIdx.x; k<D; k+=blockDim.x) {
    const float q = Q[kx][k];
      
    // coalesced memory read
    for (int n=0; n<6; n++) {
      ei[n] = E[ix][n][k] * q;
      ej[n] = E[jx][n][k];
    }

    // block EEt
    for (int n=0; n<6; n++) {
      for (int m=0; m<6; m++) {
        dS[n][m] += ei[n] * ej[m];
      }
    }
  }

  __syncthreads();
  __shared__ float sdata[THREADS];

  for (int n=0; n<6; n++) {
    for (int m=0; m<6; m++) {
      sdata[threadIdx.x] = dS[n][m];

      blockReduce(sdata);

      if (threadIdx.x == 0) {
        S[blockIdx.x][n][m] = sdata[0];
      }
    }
  }
}


__global__ void Ev6x1_kernel(
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> E,
    const torch::PackedTensorAccessor32<float, 2,torch::RestrictPtrTraits> Q,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> w,
    const torch::PackedTensorAccessor32<long,2,torch::RestrictPtrTraits> idx,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> v)
{
  const int D = E.size(2);
  const int kx = idx[blockIdx.x][0];

  float b[6];
  for (int n=0; n<6; n++) {
    b[n] = 0.0;
  }

  for (int k=threadIdx.x; k<D; k+=blockDim.x) {
    const float q_w = Q[kx][k] * w[kx][k];

    for (int n=0; n<6; n++) {
      b[n] += q_w * E[blockIdx.x][n][k];
    }
  }

  __syncthreads();
  __shared__ float sdata[THREADS];

  for (int n=0; n<6; n++) {
    sdata[threadIdx.x] = b[n];
    blockReduce(sdata);

    if (threadIdx.x == 0) {
      v[blockIdx.x][n] += sdata[0];
    }
  }
}

// EvT6x1_kernel<<<ix.size(0), THREADS>>>(
__global__ void EvT6x1_kernel(
  const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> E,
  const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> x,
  const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> idx,
  torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> w)
{

  // blockIdx.x = index of the block, which in this case has size N+ii.size(0) (aka number of keyframes + number of measurements)

  const int HW = E.size(2);
  const int ix = idx[blockIdx.x];

  if (idx[blockIdx.x] <= 0 || idx[blockIdx.x] >= x.size(0))
    return;

  // Here, ix and block.Idx.x are fixed.
  // threadIdx.x = index of the thread in the block
  // blockDim.x = number of threads per block (aka 256 here)
  for (int k=threadIdx.x; k<HW; k+=blockDim.x) { 
    float dw = 0;
    for (int n=0; n<6; n++) { // D=6 for SE(3) poses
      dw += E[blockIdx.x][n][k] * x[ix][n]; // Et * delta_x
    }
    w[blockIdx.x][k] = dw;
  }
}

class SparseBlock {
  public:

    Eigen::SparseMatrix<double> A;
    Eigen::VectorX<double> b;

    SparseBlock(int N, int M) : N(N), M(M) {
      A = Eigen::SparseMatrix<double>(N*M, N*M);
      b = Eigen::VectorXd::Zero(N*M);
    }

    SparseBlock(Eigen::SparseMatrix<double> const& A, Eigen::VectorX<double> const& b, 
        int N, int M) : A(A), b(b), N(N), M(M) {}

    void (torch::Tensor As, torch::Tensor ii, torch::Tensor jj) {
      auto As_cpu = As.to(torch::kCPU).to(torch::kFloat64);
      auto ii_cpu = ii.to(torch::kCPU).to(torch::kInt64);
      auto jj_cpu = jj.to(torch::kCPU).to(torch::kInt64);

      // An accessor allows you to index (andupdate_lhs access) the tensor...
      auto As_accessor = As_cpu.accessor<double,3>();
      auto ii_accessor = ii_cpu.accessor<long,1>();
      auto jj_accessor = jj_cpu.accessor<long,1>();

      // Build Sparse Matrix from triplets (col,row,value)
      // Builds Sparse Hessian from Dense Hessian
      std::vector<T> tripletList;
      for (int n = 0; n < ii.size(0); n++) {
        const int i = ii_accessor[n];
        const int j = jj_accessor[n];

        if (i >= 0 && j >= 0) { // this is because the (i,j) that are negative are the ones that are not optimized over (aka fixed poses)
          for (int k = 0; k < M; k++) { // M=6
            for (int l = 0; l < M; l++) { // M=6
              double val = As_accessor[n][k][l];
              // (col, row, value)
              tripletList.push_back(T(M * i + k, M * j + l, val));
            }
          }
        }
      }
      A.setFromTriplets(tripletList.begin(), tripletList.end());
    }

    void update_rhs(torch::Tensor bs, torch::Tensor ii) {
      auto bs_cpu = bs.to(torch::kCPU).to(torch::kFloat64);
      auto ii_cpu = ii.to(torch::kCPU).to(torch::kInt64);

      auto bs_accessor = bs_cpu.accessor<double,2>();
      auto ii_accessor = ii_cpu.accessor<long,1>();

      for (int n=0; n<ii.size(0); n++) {
        const int i = ii_accessor[n];
        if (i >= 0) {
          for (int j=0; j<M; j++) {
            b(i*M + j) += bs_accessor[n][j];
          }
        }
      }
    }

    SparseBlock operator-(const SparseBlock& S) {
      return SparseBlock(A - S.A, b - S.b, N, M);
    }

    std::tuple<torch::Tensor, torch::Tensor> get_dense() {
      Eigen::MatrixXd Ad = Eigen::MatrixXd(A);

      torch::Tensor H = torch::from_blob(Ad.data(), {N*M, N*M}, torch::TensorOptions()
        .dtype(torch::kFloat64)).to(torch::kCUDA).to(torch::kFloat32);

      torch::Tensor v = torch::from_blob(b.data(), {N*M, 1}, torch::TensorOptions()
        .dtype(torch::kFloat64)).to(torch::kCUDA).to(torch::kFloat32);

      return std::make_tuple(H, v);

    }

    torch::Tensor solve(const float lm=0.0001, const float ep=0.1) {

      torch::Tensor dx;

      Eigen::SparseMatrix<double> L(A);
      L.diagonal().array() += ep + lm * L.diagonal().array();

      Eigen::SimplicialLLT<Eigen::SparseMatrix<double>> solver;
      solver.compute(L);

      if (solver.info() == Eigen::Success) {
        Eigen::VectorXd x = solver.solve(b);
        dx = torch::from_blob(x.data(), {N, M}, torch::TensorOptions()
          .dtype(torch::kFloat64)).to(torch::kCUDA).to(torch::kFloat32);
      }
      else {
        printf("ERROR solving Cholesky!!\n");
        dx = torch::zeros({N, M}, torch::TensorOptions()
          .device(torch::kCUDA).dtype(torch::kFloat32));
      }
      
      return dx;
    }

  private:
    const int N;
    const int M;

};


SparseBlock schur_block(torch::Tensor E,
                        torch::Tensor Q,
                        torch::Tensor w,
                        torch::Tensor ii,
                        torch::Tensor jj,
                        torch::Tensor kk,
                        const int kf0,
                        const int kf1)
{

  torch::Tensor ii_cpu = ii.to(torch::kCPU);
  torch::Tensor jj_cpu = jj.to(torch::kCPU);
  torch::Tensor kk_cpu = kk.to(torch::kCPU);

  const int P = kf1 - kf0;
  const long* ii_data = ii_cpu.data_ptr<long>();
  const long* jj_data = jj_cpu.data_ptr<long>();
  const long* kk_data = kk_cpu.data_ptr<long>();

  std::vector<std::vector<long>> graph(P);
  std::vector<std::vector<long>> index(P);

  for (int n=0; n<ii_cpu.size(0); n++) {
    const int j = jj_data[n];
    const int k = kk_data[n];

    if (j >= kf0 && j <= kf1) {
      const int t = j - kf0;
      graph[t].push_back(k);
      index[t].push_back(n);
    }
  }

  std::vector<long> ii_list, jj_list, idx, jdx;

  for (int i=0; i<P; i++) {
    for (int j=0; j<P; j++) {
      for (int k=0; k < graph[i].size(); k++) {
        for (int l=0; l < graph[j].size(); l++) {
          if (graph[i][k] == graph[j][l]) {
            ii_list.push_back(i);
            jj_list.push_back(j);

            idx.push_back(index[i][k]);
            idx.push_back(index[j][l]);
            idx.push_back(graph[i][k]);
          }
        }
      }
    }
  }

  torch::Tensor ix_cuda = torch::from_blob(idx.data(), {long(idx.size())}, 
    torch::TensorOptions().dtype(torch::kInt64)).to(torch::kCUDA).view({-1, 3});

  torch::Tensor jx_cuda = torch::stack({kk_cpu}, -1)
    .to(torch::kCUDA).to(torch::kInt64);

  torch::Tensor ii2_cpu = torch::from_blob(ii_list.data(), {long(ii_list.size())}, 
    torch::TensorOptions().dtype(torch::kInt64)).view({-1});

  torch::Tensor jj2_cpu = torch::from_blob(jj_list.data(), {long(jj_list.size())}, 
    torch::TensorOptions().dtype(torch::kInt64)).view({-1});

  torch::Tensor S = torch::zeros({ix_cuda.size(0), 6, 6}, 
    torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));

  torch::Tensor v = torch::zeros({jx_cuda.size(0), 6},
    torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));

  EEt6x6_kernel<<<ix_cuda.size(0), THREADS>>>(
    E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    Q.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    ix_cuda.packed_accessor32<long,2,torch::RestrictPtrTraits>(),
    S.packed_accessor32<float,3,torch::RestrictPtrTraits>());

  Ev6x1_kernel<<<jx_cuda.size(0), THREADS>>>(
    E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    Q.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    w.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    jx_cuda.packed_accessor32<long,2,torch::RestrictPtrTraits>(),
    v.packed_accessor32<float,2,torch::RestrictPtrTraits>());

  // schur block
  SparseBlock A(P, 6);
  A.update_lhs(S, ii2_cpu, jj2_cpu);
  A.update_rhs(v, jj_cpu - kf0);

  return A;
}


std::vector<torch::Tensor> ba_cuda(
    torch::Tensor poses,
    torch::Tensor body_poses, // SEND IDENTITY, this is not supposed to work otw
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor extrinsics,
    torch::Tensor disps_sens,
    torch::Tensor targets,
    torch::Tensor weights,
    torch::Tensor eta,
    torch::Tensor ii,
    torch::Tensor jj,
    const int kf0,
    const int kf1,
    const int iterations,
    const float lm,
    const float ep,
    const bool motion_only)
{
  auto opts = poses.options();
  const int num = ii.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  torch::Tensor ts = torch::arange(kf0, kf1).to(torch::kCUDA);
  torch::Tensor ii_expanded = torch::cat({ts, ii}, 0);
  torch::Tensor jj_expanded = torch::cat({ts, jj}, 0);

  std::tuple<torch::Tensor, torch::Tensor> ii_unique = 
    torch::_unique(ii_expanded, true, true);

  torch::Tensor kx = std::get<0>(ii_unique);
  torch::Tensor kk_exp = std::get<1>(ii_unique);
    
  torch::Tensor dx;
  torch::Tensor dz;

  // initialize buffers
  torch::Tensor Hs = torch::zeros({4, num, 6, 6}, opts);
  torch::Tensor vs = torch::zeros({2, num, 6}, opts);
  torch::Tensor Eii = torch::zeros({num, 6, ht*wd}, opts);
  torch::Tensor Eij = torch::zeros({num, 6, ht*wd}, opts);
  torch::Tensor Cii = torch::zeros({num, ht*wd}, opts);
  torch::Tensor wi = torch::zeros({num, ht*wd}, opts);

  for (int itr=0; itr<iterations; itr++) {

    projective_transform_kernel<<<num, THREADS>>>(
      targets.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
      weights.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
      poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
      body_poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
      disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
      extrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
      ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
      jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
      Hs.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
      vs.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      Eii.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      Eij.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      Cii.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
      wi.packed_accessor32<float,2,torch::RestrictPtrTraits>());


    // pose x pose block
    SparseBlock A(kf1 - kf0, 6); // (N, M=6) -> A = (N*M, N*M), b = (N*M)

    A.update_lhs(Hs.reshape({-1, 6, 6}), 
        torch::cat({ii, ii, jj, jj}) - kf0, 
        torch::cat({ii, jj, ii, jj}) - kf0);

    A.update_rhs(vs.reshape({-1, 6}), 
        torch::cat({ii, jj}) - kf0);

    if (motion_only) {
      dx = A.solve(lm, ep);

      // update poses
      pose_retr_kernel<<<1, THREADS>>>(
        poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(), kf0, kf1);
    } else {
      // add depth residual if there are depth sensor measurements
      const float alpha = 0.05; // THIS should be a huber weight per measurement, not a constant
      torch::Tensor m = (disps_sens.index({kx, "..."}) > 0).to(torch::TensorOptions().dtype(torch::kFloat32)).view({-1, ht*wd});
      torch::Tensor C = accum_cuda(Cii, ii, kx) + m * alpha + (1 - m) * eta.view({-1, ht*wd}); 
      torch::Tensor w = accum_cuda(wi, ii, kx) - m * alpha * (disps.index({kx, "..."}) - disps_sens.index({kx, "..."})).view({-1, ht*wd});
      torch::Tensor Q = 1.0 / C; 

      torch::Tensor Ei = accum_cuda(Eii.view({num, 6*ht*wd}), ii, ts).view({kf1-kf0, 6, ht*wd});
      torch::Tensor E = torch::cat({Ei, Eij}, 0);

      SparseBlock S = schur_block(E, Q, w, ii_expanded, jj_expanded, kk_exp, kf0, kf1);

      // Return (A-S), the reduced camera matrix, and solve it via GTSAM together with IMU

      dx = (A - S).solve(lm, ep);

      // Given dx, solve for dz

      torch::Tensor ix = jj_expanded - kf0;
      torch::Tensor dw = torch::zeros({ix.size(0), ht*wd}, opts);

      EvT6x1_kernel<<<ix.size(0), THREADS>>>(
        E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
        dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        ix.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
        dw.packed_accessor32<float,2,torch::RestrictPtrTraits>());

      dz = Q * (w - accum_cuda(dw, ii_expanded, kx));

      // update poses
      pose_retr_kernel<<<1, THREADS>>>(
        poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(), kf0, kf1);

      // update disparity maps
      disp_retr_kernel<<<kx.size(0), THREADS>>>(
        disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
        dz.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        kx.packed_accessor32<long,1,torch::RestrictPtrTraits>());
    }

  }

  return {dx, dz};
}



torch::Tensor frame_distance_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor ii,
    torch::Tensor jj,
    const float beta)
{
  auto opts = poses.options();
  const int num = ii.size(0);

  torch::Tensor dist = torch::zeros({num}, opts);

  frame_distance_kernel<<<num, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    dist.packed_accessor32<float,1,torch::RestrictPtrTraits>(), beta);

  return dist;
}


std::vector<torch::Tensor> projmap_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor ii,
    torch::Tensor jj)
{
  auto opts = poses.options();
  const int num = ii.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  torch::Tensor coords = torch::zeros({num, ht, wd, 3}, opts);
  torch::Tensor valid = torch::zeros({num, ht, wd, 1}, opts);

  projmap_kernel<<<num, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    coords.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    valid.packed_accessor32<float,4,torch::RestrictPtrTraits>());

  return {coords, valid};
}


torch::Tensor depth_filter_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor ix,
    torch::Tensor thresh)
{
  const int num = ix.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  torch::Tensor counter = torch::zeros({num, ht, wd}, disps.options());

  dim3 blocks(num, 6, NUM_BLOCKS(ht * wd));

  depth_filter_kernel<<<blocks, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    ix.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    thresh.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    counter.packed_accessor32<float,3,torch::RestrictPtrTraits>());

  return counter;
}


torch::Tensor iproj_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics)
{

  const int nm = disps.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  auto opts = disps.options();
  torch::Tensor points = torch::zeros({nm, ht, wd, 3}, opts);

  dim3 blocks(nm, NUM_BLOCKS(ht * wd));

  iproj_kernel<<<blocks, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    points.packed_accessor32<float,4,torch::RestrictPtrTraits>());

  return points;

}


// Computes reduced camera matrix:  H_c/H_p
// Aka the schur complement of the cameras wrt points
// Eigen::MatrixXd is a dynamic 2D matrix of doubles.
std::vector<torch::Tensor>
reduced_camera_matrix_cuda(
    torch::Tensor poses,
    torch::Tensor body_poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor extrinsics,
    torch::Tensor disps_sens,
    torch::Tensor targets,
    torch::Tensor weights,
    torch::Tensor eta,
    torch::Tensor ii, // Contains active AND inactive keyframes
    torch::Tensor jj, // Contains active AND inactive keyframes
    const int kf0,    // Min active keyframe
    const int kf1)    // Max active keyframe
{
  auto opts = poses.options();
  const int M = ii.size(0); // number of measurements
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  torch::Tensor ts = torch::arange(kf0, kf1).to(torch::kCUDA);
  torch::Tensor ii_expanded = torch::cat({ts, ii}, 0);
  torch::Tensor jj_expanded = torch::cat({ts, jj}, 0);

  std::tuple<torch::Tensor, torch::Tensor> ii_unique = 
    torch::_unique(ii_expanded, true, true);

  torch::Tensor ii_kf_ids = std::get<0>(ii_unique);
  torch::Tensor kk_exp = std::get<1>(ii_unique); // same size as ii_expanded, ii_expanded[i] == ii_kf_ids[kk_exp[i]]
    
  // initialize buffers
  torch::Tensor Hs = torch::zeros({4, M, 6, 6}, opts);
  torch::Tensor vs = torch::zeros({2, M, 6}, opts);
  torch::Tensor Eiz = torch::zeros({M, 6, ht*wd}, opts);
  torch::Tensor Ejz = torch::zeros({M, 6, ht*wd}, opts);
  torch::Tensor Cii = torch::zeros({M, ht*wd}, opts);
  torch::Tensor wi = torch::zeros({M, ht*wd}, opts);

  // Here we should iterate...
  // By iterating on the whole function, we are re-allocating buffers all the time...
  projective_transform_kernel<<<M, THREADS>>>(
    targets.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    weights.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    body_poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    extrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    Hs.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    vs.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    Eiz.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    Ejz.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    Cii.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    wi.packed_accessor32<float,2,torch::RestrictPtrTraits>());

  // pose x pose block
  SparseBlock A(kf1 - kf0, 6); // (N, M=6) -> A = (N*M, N*M), b = (N*M)

  A.update_lhs(Hs.reshape({-1, 6, 6}), 
      torch::cat({ii, ii, jj, jj}) - kf0, 
      torch::cat({ii, jj, ii, jj}) - kf0);

  A.update_rhs(vs.reshape({-1, 6}), 
      torch::cat({ii, jj}) - kf0);

  // add depth residual if there are depth sensor measurements
  const float alpha = 0.05;
  torch::Tensor m = (disps_sens.index({ii_kf_ids, "..."}) > 0).to(torch::TensorOptions().dtype(torch::kFloat32)).view({-1, ht*wd});
  torch::Tensor C = accum_cuda(Cii, ii, ii_kf_ids) + m * alpha + (1 - m) * eta.view({-1, ht*wd}); // add alpha if sensed depth, eta if not.
  torch::Tensor w = accum_cuda(wi, ii, ii_kf_ids)  - m * alpha * (disps.index({ii_kf_ids, "..."}) - disps_sens.index({ii_kf_ids, "..."})).view({-1, ht*wd});
  torch::Tensor Q = 1.0 / C;

  // Accumulate all contributions from the different flows to get the Eii block, that constraints the i-th camera depth
  torch::Tensor Ei = accum_cuda(Eiz.view({M, 6*ht*wd}), ii, ts).view({kf1-kf0, 6, ht*wd}); 
  torch::Tensor E = torch::cat({Ei, Ejz}, 0);

  SparseBlock S = schur_block(E, Q, w, ii_expanded, jj_expanded, kk_exp, kf0, kf1);

  // Return (A-S), the reduced camera matrix, and solve it via GTSAM together with IMU
  SparseBlock rcm = (A - S);//dx = (A - S).solve(lm, ep);
  // We also want the inverse of rcm! Because it is our marginal covariance!
  // But we rather would take the covariance of the rcm with the IMU...
  auto tuple = rcm.get_dense();
  return {std::get<0>(tuple), std::get<1>(tuple), Q, E, w};
}

// Given dx, calc dz.
// With both dx and dz, retract poses and depths.
void solve_depth_cuda(
    torch::Tensor dx,
    torch::Tensor disps,
    torch::Tensor Q,
    torch::Tensor E,
    torch::Tensor w,
    torch::Tensor ii,
    torch::Tensor jj,
    const int kf0,
    const int kf1)
{
    auto opts = disps.options();
    const int ht = disps.size(1);
    const int wd = disps.size(2);

    // Given dx, solve for dz
    torch::Tensor dz;


    torch::Tensor ts = torch::arange(kf0, kf1).to(torch::kCUDA);
    torch::Tensor ii_expanded = torch::cat({ts, ii}, 0);
    torch::Tensor jj_expanded = torch::cat({ts, jj}, 0); // expanded because it contains ii->jj, prepended with 0->0,...N->N, aka self-loops, because E, is cat(Eii, Eij)

    std::tuple<torch::Tensor, torch::Tensor> ii_unique = 
      torch::_unique(ii_expanded, true, true);

    torch::Tensor ii_kf_ids = std::get<0>(ii_unique);
    torch::Tensor kk_exp = std::get<1>(ii_unique);

    torch::Tensor ix = jj_expanded - kf0; // Size of P + ii.size(0), aka P + N
    torch::Tensor dw = torch::zeros({ix.size(0), ht*wd}, opts);

    //  <<<numBlocks, threadsPerBlock>>>
    // THREADS = 256
    EvT6x1_kernel<<<ix.size(0), THREADS>>>(
      E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
      ix.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
      dw.packed_accessor32<float,2,torch::RestrictPtrTraits>());

    dz = Q * (w - accum_cuda(dw, ii_expanded, ii_kf_ids));

    // TODO: remove since gtsam does the retraction itself...
    // update poses
    // pose_retr_kernel<<<1, THREADS>>>(
    //   poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    //   droid_dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(), kf0, kf1);

    // update disparity maps
    disp_retr_kernel<<<ii_kf_ids.size(0), THREADS>>>(
      disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      dz.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
      ii_kf_ids.packed_accessor32<long,1,torch::RestrictPtrTraits>());
}

// void solve_cuda(
//     torch::Tensor A,
//     torch::Tensor S,
//     const float lm,
//     const float ep)
// {
//   dx = (A - S).solve(lm, ep); // Unfortunately, A & S are SchurBlocks, not tensors...
//   return dx;
// }
// 
void solve_poses_cuda(
    torch::Tensor poses,
    torch::Tensor dx,
    const int kf0,
    const int kf1)
{
  pose_retr_kernel<<<1, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    kf0, kf1);
}
